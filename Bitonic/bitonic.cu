#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


// Your job is to implemment a bitonic sort. A description of the bitonic sort
// can be see at:
// http://en.wikipedia.org/wiki/Bitonic_sort
    
__device__
    void compare(float *data, int pos1, int pos2){
    if(data[pos1] > data[pos2]){
        float temp = data[pos1];
        data[pos1] = data[pos2];
        data[pos2] = temp;
    }
}
__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid  = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();
    if(tid < 32)
    for (int stage = 1; stage <= 6; stage++)
    {
        //MERGE
        int n = (int) pow((float)2,(float)stage);
        int group = (2*tid)/n;
        int i = tid%(n/2);
        compare(sdata,n*group+i, n*group+n-i-1);
        __syncthreads();
        for (int substage = stage -1; substage > 0; substage--)
        {
            int n = (int) pow((float)2,(float)substage);
            int group = (2*tid)/n;
            int i = tid%(n/2);
            compare(sdata,n*group+i, n*group+i+n/2);
                
        }
        
    }
    __syncthreads();
    d_out[tid] = sdata[tid];
}

int compareFloat (const void * a, const void * b)
{
  if ( *(float*)a <  *(float*)b ) return -1;
  if ( *(float*)a == *(float*)b ) return 0;
  if ( *(float*)a >  *(float*)b ) return 1;
  return 0;                     // should never reach this
}

void printArray(float* array, int n){
	printf("\n");
	for(int i = 0; i < n; i++){
		printf("%f - ",array[i]);
	}
}


int main(int argc, char **argv)
{
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // generate the input array on the host
    float h_in[ARRAY_SIZE];
    float h_sorted[ARRAY_SIZE];
    float h_out[ARRAY_SIZE];
    for(int i = 0; i < ARRAY_SIZE; i++) {
        // generate random float in [0, 1]
        h_in[i] = (float)random()/(float)RAND_MAX;
        h_sorted[i] = h_in[i];
    }
    qsort(h_sorted, ARRAY_SIZE, sizeof(float), compareFloat);

    // declare GPU memory pointers
    float * d_in, * d_out;

    // allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);

    // transfer the input array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice); 
    printArray(h_in, ARRAY_SIZE);
    batcherBitonicMergesort64<<<1, ARRAY_SIZE, ARRAY_SIZE * sizeof(float)>>>(d_out, d_in);
    
    // copy back the sum from GPU
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
    printArray(h_out, ARRAY_SIZE);
    
  
    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out);
}
